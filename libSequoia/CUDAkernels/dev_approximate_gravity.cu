#include "hip/hip_runtime.h"
#include "support_kernels.cu"
#include "dev_shared_traverse_functions.cu"

#include <stdio.h>

//Some settings

#define TEXTURES
#define OLDPREFIX
#define DOGRAV

#if 1      /* use this if you want the potential to be in double precision */
#define _GPOTDP_
#endif


struct ds64
{
  float2 val;
  __host__ __device__ ds64() {}
  __host__ __device__ ds64(float x) : val(make_float2(x, x)) {}
  __host__ __device__ ds64 operator+=(const float x) 
  {
    const float vx = val.x + x;
    const float vy = val.y - ((vx - val.x) - x);
    val = make_float2(vx, vy);
    return *this;
  }
  __host__ __device__ double to_double() const { return (double)val.x + (double)val.y; }
  __host__ __device__ float to_float() const { return (float)((double)val.x + (double)val.y);}
};

struct devForce
{
  ds64 ax, ay, az;   // 6
#ifdef _GPOTDP_
  ds64 pot;          // 8
#else
  float pot;         // 7
  int  iPad;         // 8
#endif
  __host__ __device__ devForce() {}
  __device__ devForce(const float v) : ax(v), ay(v), az(v), pot(v) {}
  __device__ float4 to_float4() 
  {
#ifdef _GPOTDP_
    return (float4){ax.to_float(), ay.to_float(), az.to_float(), pot.to_float()};
#else
    return (float4){ax.to_float(), ay.to_float(), az.to_float(), pot};
#endif
  }
};


texture<float4, 1, hipReadModeElementType> texNodeSize;
texture<float4, 1, hipReadModeElementType> texNodeCenter;
texture<float4, 1, hipReadModeElementType> texMultipole;
texture<float4, 1, hipReadModeElementType> texBody;


__device__ devForce body_body(
    devForce iforce,
    const float4 posi,
    const float4 _hi,
    const float  massj,
    const float3 posj,
    const float3 _hj,
    const bool   not_selfGravity)
{
  const float3 dr = {posj.x - posi.x, posj.y - posi.y, posj.z - posi.z};
  const float  r2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

  const float  rinv  = rsqrtf(r2);
  const float  rinv2 = rinv   * rinv;
  const float mrinv1 = rinv   * massj;
  const float mrinv3 = rinv2  * mrinv1;
  
  const float hi    = _hi.x;
  const float invhi = _hi.y;
  const float hj    = _hj.x;
  const float invhj = _hj.y;
  const float hflag = _hj.z;

  const float h2i = hi*hi;
  const float h2j = hj*hj;

  if (r2 > fmaxf(h2i, h2j))
  {
    iforce.ax  +=   mrinv3 * dr.x;
    iforce.ay  +=   mrinv3 * dr.y;
    iforce.az  +=   mrinv3 * dr.z;
    iforce.pot += (-mrinv1);
  } 
  else
  {
    const float2 invq  = not_selfGravity ? (float2){rinv * hi,   rinv * hj  } : (float2){0.0f, 0.0f};
    const float2 q     = not_selfGravity ? (float2){1.0f/invq.x, 1.0f/invq.y} : (float2){0.0f, 0.0f};
    const float2 q2    = {q.x*q .x, q.y*q. y};
    const float2 q3    = {q.x*q2.x, q.y*q2.y};
    const float2 invq2 = {invq.x*invq. x, invq.y*invq .y};
    const float2 invq3 = {invq.x*invq2.x, invq.y*invq2.y};
    
    const float2 f   = {q.x < 0.5f, q.y < 0.5f};
    const float2 acc = 
    {
        (       f.x) * (10.666666666667f + q2.x * (32.0f * q.x + (-38.4f))) + 
        (1.0f - f.x) * (21.333333333333f + (-48.0f)*q.x + 38.4f*q2.x + (-10.666666666667f)*q3.x + (-0.066666666667f) * invq3.x),
        (       f.y) * (10.666666666667f + q2.y * (32.0f * q.y + (-38.4f))) + 
        (1.0f - f.y) * (21.333333333333f + (-48.0f)*q.y + 38.4f*q2.y + (-10.666666666667f)*q3.y + (-0.066666666667f) * invq3.y)
    };
    
    const float2 pot =
    {
        (       f.x) * ((-2.8f) + q2.x * (5.333333333333f  + q2.x * (6.4f * q.x + (-9.6f)))) + 
        (1.0f - f.x) * ((-3.2f) + 0.066666666667f * invq.x + q2.x * (10.666666666667f + q.x * ((-16.0f) + q.x * (9.6f + (-2.133333333333f) * q.x)))),
        (       f.y) * ((-2.8f) + q2.y * (5.333333333333f  + q2.y * (6.4f * q.y + (-9.6f)))) + 
        (1.0f - f.y) * ((-3.2f) + 0.066666666667f * invq.y + q2.y * (10.666666666667f + q.y * ((-16.0f) + q.y * (9.6f + (-2.133333333333f) * q.y)))),
    };
     
    const float2 mj1 = {massj * invhi,       massj * invhj };
    const float2 mj2 = {mj1.x * invhi*invhi, mj1.y * invhj*invhj};

    const float2 g = {r2 <= h2i, r2 <= h2j};
    const float gacc = not_selfGravity ?
      0.5f*(g.x*mj2.x*acc.x + (1.0f - g.x)*mrinv3 + 
            g.y*mj2.y*acc.y + (1.0f - g.y)*mrinv3) : 0.0f;
    const float gpot = not_selfGravity ?
      0.5f*(g.x*mj1.x*pot.x + (g.x - 1.0f)*mrinv1 + 
            g.y*mj1.y*pot.y + (g.y - 1.0f)*mrinv1) : (-1.4f)*hflag*(mj1.x + mj1.y);


    iforce.ax  += gacc * dr.x;
    iforce.ay  += gacc * dr.y;
    iforce.az  += gacc * dr.z;
    iforce.pot += gpot;
  }

  return iforce;
}

__device__ float4 get_D04(const float ds2)
{
  const float ids = rsqrt(ds2);
  const float ids2 = ids *ids;
  const float ids3 = ids2*ids;  
  const float ids5 = ids3*ids2;
  const float ids7 = ids5*ids2;
  return (float4){+ids, -ids3, +3.0f*ids5, -15.0f*ids7};
} 

__device__ devForce body_node(
    devForce acc, 
    const float4 pos,
    const float mass, 
    const float3 com,
    const float3 Q0,  
    const float3 Q1)
{
  const float3  dr = {pos.x - com.x, pos.y - com.y, pos.z - com.z};
  const float   r2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
  const float4 D04 = get_D04(r2);

  const float  D0  = D04.x*mass;
  const float  D1  = D04.y*mass;
  const float  D2  = D04.z*mass;
  const float  D3  = D04.w*mass;

  const float oct_q11 = Q0.x;
  const float oct_q22 = Q0.y;
  const float oct_q33 = Q0.z;
  const float oct_q12 = Q1.x;
  const float oct_q13 = Q1.y;
  const float oct_q23 = Q1.z;

  const float Qii = oct_q11 + oct_q22 + oct_q33;
  const float QijRiRj =
    (oct_q11*dr.x*dr.x + oct_q22*dr.y*dr.y + oct_q33*dr.z*dr.z) +
    2.0f*(oct_q12*dr.y*dr.x + oct_q13*dr.z*dr.x + oct_q23*dr.y*dr.z);

  const float C01a = D1 + 0.5f*D2*Qii + 0.5f*D3*QijRiRj;
  acc.pot   +=     -(D0 + 0.5f*D1*Qii + 0.5f*D2*QijRiRj);
  acc.ax    += C01a*dr.x + D2*(oct_q11*dr.x + oct_q12*dr.y + oct_q13*dr.z);
  acc.ay    += C01a*dr.y + D2*(oct_q12*dr.x + oct_q22*dr.y + oct_q23*dr.z);
  acc.az    += C01a*dr.z + D2*(oct_q13*dr.x + oct_q23*dr.y + oct_q33*dr.z);

  return acc;
}

template<int DIM2, int SHIFT>
__device__ float4 approximate_gravity(
    int DIM2x, int DIM2y,
    int tid, 
    int tx, int ty,
    int    body_i, 
    float4 pos_i,
    float4 h_i,
    real4 group_pos,
    float eps2,
    uint2 node_begend,
    real4 *multipole_data,
    real4 *body_pos,
    int *shmem,
    int *lmem,

    int &apprCount, int &direCount,
    volatile float4 *boxSizeInfo,
    float4 groupSize,
    volatile float4 *boxCenterInfo,
    float group_eps,

    real4 *body_h,
    int *bodies_IDs) 
{
  devForce acc_i(0.0f);

  /*********** set necessary thread constants **********/

  const int DIMx = 1  << DIM2x;
  const int DIMy = 1  << DIM2y;
  const int DIM  = 1  << DIM2;
  const int offs = ty << DIM2x;

  /*********** shared memory distribution **********/

  //  begin,    end,   size
  // -----------------------
  int *approx = (int*)&shmem [     0];            //  0*DIM,  2*DIM,  2*DIM
  int *direct = (int*)&approx[ 2*DIM];            //  2*DIM,  3*DIM,  1*DIM
  int *nodes  = (int*)&direct[   DIM];            //  3*DIM, 13*DIM, 10*DIM
  int *prefix = (int*)&nodes [10*DIM];            // 13*DIM, 15*DIM,  2*DIM

  float  *node_mon0 = (float* )&nodes    [DIM];   //  4*DIM,  5*DIM,  1*DIM
  float3 *node_mon1 = (float3*)&node_mon0[DIM];   //  5*DIM,  8*DIM,  3*DIM
  float3 *node_oct0 = (float3*)&node_mon1[DIM];   //  8*DIM, 11*DIM,  3*DIM
  float3 *node_oct1 = (float3*)&node_oct0[DIM];   // 11*DIM, 14*DIM,  3*DIM

  int    *body_list = (int*   )&nodes    [  DIM]; //  4*DIM,  8*DIM,  4*DIM
  float  *sh_mass   = (float* )&body_list[4*DIM]; //  8*DIM,  9*DIM,  1*DIM
  float3 *sh_pos    = (float3*)&sh_mass  [  DIM]; //  9*DIM, 12*DIM   3*DIM
  int    *sh_jid    = (int*   )&sh_pos   [  DIM]; // 12*DIM, 13*DIM,  1*DIM
  float3 *sh_h      = (float3*)&sh_jid   [3*DIM]; // 15*DIM, 18*DIM,  3*DIM

  devForce *shForce = (devForce*)shmem;

  /*********** stack **********/

  int *nstack = lmem;

  /*********** begin tree-walk **********/

  int n_approx = 0;
  int n_direct = 0;

  for (int root_node = node_begend.x; root_node < node_begend.y; root_node += DIM) 
  {
    int n_nodes0 = min(node_begend.y - root_node, DIM);
    int n_stack0 = 0;
    int n_stack_pre = 0;

    { 
      nstack[ACCS<SHIFT>(n_stack0)] = root_node + tid;   
      n_stack0++; 
    }

    /*********** walk each level **********/
    while (n_nodes0 > 0) 
    {
      int n_nodes1 = 0;
      int n_offset = 0;

      int n_stack1 = n_stack0;
      int c_stack0 = n_stack_pre;

      /*********** walk a level **********/
      while(c_stack0 < n_stack0) 
      {
        /***
         **** --> fetch the list of nodes rom LMEM
         ***/
        bool use_node = tid <  n_nodes0;
        { 
          prefix[tid] = nstack[ACCS<SHIFT>(c_stack0)];   
          c_stack0++; 
        }
        __syncthreads();
        int node  = prefix[min(tid, n_nodes0 - 1)];

        if(n_nodes0 > 0)       //Work around pre 4.1 compiler bug
          n_nodes0 -= DIM;

        /***
         **** --> process each of the nodes in the list in parallel
         ***/

#ifndef TEXTURES
        float4 nodeSize = get_float4(boxSizeInfo[node]);                   //Fetch the size of the box. Size.w = child info
        float4 node_pos = get_float4(boxCenterInfo[node]);                 //Fetch the center of the box. center.w = opening info
#else
        float4 nodeSize =  tex1Dfetch(texNodeSize, node);
        float4 node_pos =  tex1Dfetch(texNodeCenter, node);
#endif

        int node_data = __float_as_int(nodeSize.w);

        //Check if a cell has to be opened
#if 0 // def IMPBH
        //Improved barnes hut method

#ifndef TEXTURES
        float4 nodeCOM = multipole_data[node*3];
#else
        float4 nodeCOM = tex1Dfetch(texMultipole,node*3);
#endif  

        nodeCOM.w      = node_pos.w;
        bool   split   = split_node_grav_impbh(nodeCOM, group_pos, groupSize);
#else   /* IMPBH */
        //Minimum distance method
        bool   split   = split_node_grav_md(node_pos, nodeSize, group_pos, groupSize);  //Check if node should be split
#endif /* IMPBH */

        bool leaf       = node_pos.w <= 0;  //Small AND equal incase of a 1 particle cell       //Check if it is a leaf
        //         split = true;


        uint mask    = BTEST((split && !leaf) && use_node);               // mask = #FFFFFFFF if use_node+split+not_a_leaf==true, otherwise zero
        int child    =    node_data & 0x0FFFFFFF;                         //Index to the first child of the node
        int nchild   = (((node_data & 0xF0000000) >> 28)) & mask;         //The number of children this node has


        /***
         **** --> calculate prefix
         ***/

        int *prefix0 = &prefix[  0];
        int *prefix1 = &prefix[DIM];

#ifdef OLDPREFIX
        int n_total = calc_prefix<DIM2>(prefix, tid,  nchild);
        prefix[tid] += n_offset - nchild;
        __syncthreads();
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix, nchild, tid);        // inclusive scan to compute memory offset of each child
        int n_total = prefix[blockDim.x - 1];                              // fetch total number of children, i.e. offset of the last child -1
        __syncthreads();                                                   // thread barrier to make sure that warps completed their jobs
        prefix[tid] += n_offset - nchild;                                  // convert inclusive into exclusive scan for referencing purpose
        __syncthreads();                                                   // thread barrier
#endif

        for (int i = n_offset; i < n_offset + n_total; i += DIM)         //nullify part of the array that will be filled with children
          nodes[tid + i] = 0;                                          //but do not touch those parts which has already been filled
        __syncthreads();                                                 //Thread barrier to make sure all warps finished writing data

        bool flag = (split && !leaf) && use_node;                        //Flag = use_node + split + not_a_leaf;Use only non_leaf nodes that are to be split
        if (flag) nodes[prefix[tid]] = child;                            //Thread with the node that is about to be split
        __syncthreads();                                                 //writes the first child in the array of nodes

        /*** in the following 8 lines, we calculate indexes of all the children that have to be walked from the index of the first child***/
        if (flag && nodes[prefix[tid] + 1] == 0) nodes[prefix[tid] + 1] = child + 1; __syncthreads();
        if (flag && nodes[prefix[tid] + 2] == 0) nodes[prefix[tid] + 2] = child + 2; __syncthreads();
        if (flag && nodes[prefix[tid] + 3] == 0) nodes[prefix[tid] + 3] = child + 3; __syncthreads();
        if (flag && nodes[prefix[tid] + 4] == 0) nodes[prefix[tid] + 4] = child + 4; __syncthreads();
        if (flag && nodes[prefix[tid] + 5] == 0) nodes[prefix[tid] + 5] = child + 5; __syncthreads();
        if (flag && nodes[prefix[tid] + 6] == 0) nodes[prefix[tid] + 6] = child + 6; __syncthreads();
        if (flag && nodes[prefix[tid] + 7] == 0) nodes[prefix[tid] + 7] = child + 7; __syncthreads();

        n_offset += n_total;    //Increase the offset in the array by the number of newly added nodes


        /***
         **** --> save list of nodes to LMEM
         ***/

        /*** if half of shared memory or more is filled with the the nodes, dump these into slowmem stack ***/
        while(n_offset >= DIM) 
        {
          n_offset -= DIM;
          const int offs1 = ACCS<SHIFT>(n_stack1);
          nstack[offs1] = nodes[n_offset + tid];   n_stack1++;
          n_nodes1 += DIM;

          if((n_stack1 - c_stack0) >= (LMEM_STACK_SIZE << SHIFT))
          {
            apprCount = -1; 
            return acc_i.to_float4();
          }
        }

        __syncthreads();

        /******************************/
        /******************************/
        /*****     EVALUATION     *****/
        /******************************/
        /******************************/
#if 1
        /***********************************/
        /******       APPROX          ******/
        /***********************************/

#ifdef OLDPREFIX
        n_total = calc_prefix<DIM2>(prefix, tid,  1 - (split || !use_node));
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix, 1 - (split || !use_node), tid);
        n_total = prefix[blockDim.x - 1];
#endif


        // 	n_total = calc_prefix<DIM2>(prefix, tid,  !split && use_node);         // for some unkown reason this does not work right on the GPU
        if (!split && use_node) approx[n_approx + prefix[tid] - 1] = node;
        __syncthreads();
        n_approx += n_total;

        while (n_approx >= DIM) 
        {
          n_approx -= DIM;
          int address      = (approx[n_approx + tid] << 1) + approx[n_approx + tid];
#ifndef TEXTURES
          float4 monopole  = multipole_data[address    ];
          float4 octopole0 = multipole_data[address + 1];
          float4 octopole1 = multipole_data[address + 2];
#else
          float4 monopole  = tex1Dfetch(texMultipole, address);
          float4 octopole0 = tex1Dfetch(texMultipole, address + 1);
          float4 octopole1 = tex1Dfetch(texMultipole, address + 2);
#endif

          node_mon0[tid] = monopole.w;
          node_mon1[tid] = (float3){monopole.x,  monopole.y,  monopole.z};
          node_oct0[tid] = (float3){octopole0.x, octopole0.y, octopole0.z};
          node_oct1[tid] = (float3){octopole1.x, octopole1.y, octopole1.z};

          __syncthreads();

#pragma unroll
          for (int i = 0; i < DIMx; i++)
          {
            apprCount++;
            acc_i = body_node(
                acc_i, pos_i,
                node_mon0[offs + i], node_mon1[offs + i],
                node_oct0[offs + i], node_oct1[offs + i]);
          }
          __syncthreads();
        }
        __syncthreads();
#endif

#if 1  /* DIRECT */
        /***********************************/
        /******       DIRECT          ******/
        /***********************************/

        int *sh_body = &approx[DIM];

        flag         = split && leaf && use_node;                                //flag = split + leaf + use_node
        int  jbody   = node_data & BODYMASK;                                     //the first body in the leaf
        int  nbody   = (((node_data & INVBMASK) >> LEAFBIT)+1) & BTEST(flag);    //number of bodies in the leaf masked with the flag

        body_list[tid] = direct[tid];                                            //copy list of bodies from previous pass to body_list
        sh_body  [tid] = jbody;                                                  //store the leafs first body id into shared memory

        // step 1
#ifdef OLDPREFIX
        calc_prefix<DIM2>(prefix0, tid, flag);
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix0, (int)flag, tid);       // inclusive scan on flags to construct array
#endif

        if (flag) prefix1[prefix0[tid] - 1] = tid;                             //with tidś whose leaves have to be opened
        __syncthreads();                                                      //thread barrier, make sure all warps completed the job

        // step 2
#ifdef OLDPREFIX
        int n_bodies  = calc_prefix<DIM2>(prefix0, tid, nbody);
#else
        inclusive_scan_block<ADDOP<int>, int>(prefix0, nbody, tid);        // inclusive scan to compute memory offset for each body
        int n_bodies = prefix0[blockDim.x - 1];                            //Total number of bides extract from the leaves
        __syncthreads();                                                   // thread barrier to make sure that warps completed their jobs
#endif

        direct [tid]  = prefix0[tid];                                       //Store a copy of inclusive scan in direct
        prefix0[tid] -= nbody;                                              //convert inclusive int oexclusive scan
        prefix0[tid] += 1;                                                  //add unity, since later prefix0[tid] == 0 used to check barrier

        int nl_pre = 0;                                                     //Number of leaves that have already been processed

#define NJMAX (DIM*4)
        while (n_bodies > 0) 
        {
          int nb    = min(n_bodies, NJMAX - n_direct);                    //Make sure number of bides to be extracted does not exceed
          //the amount of allocated shared memory

          // step 0                                                      //nullify part of the body_list that will be filled with bodies
          for (int i = n_direct; i < n_direct + nb; i += DIM)            //from the leaves that are being processed
            body_list[i + tid] = 0;
          __syncthreads();

          //step 1:
          if (flag && (direct[tid] <= nb) && (prefix0[tid] > 0))        //make sure that the thread indeed carries a leaf
            body_list[n_direct + prefix0[tid] - 1] = 1;                 //whose bodies will be extracted
          __syncthreads();

          //step 2:
#ifdef OLDPREFIX
          int nl = calc_prefix<DIM2>(nb, &body_list[n_direct], tid);
#else
          int nl = inclusive_scan_array<ADDOP<int>, int>              // inclusive scan to compute number of leaves to process
            (&body_list[n_direct], nb, tid);                          // to make sure that there is enough shared memory for bodies
#endif
          nb = direct[prefix1[nl_pre + nl - 1]];                        // number of bodies stored in these leaves

          // step 3:
          for (int i = n_direct; i < n_direct + nb; i += DIM) 
          {                                                              //segmented fill of the body_list
            int j = prefix1[nl_pre + body_list[i + tid] - 1];            // compute the first body in shared j-body array
            body_list[i + tid] = (i + tid - n_direct) -                 //add to the index of the first j-body in a child
              (prefix0[j] - 1) + sh_body[j];         //the index of the first child in body_list array
          }
          __syncthreads();


          /**************************************************
           *  example of what is accomplished in steps 0-4   *
           *       ---------------------------               *
           * step 0: body_list = 000000000000000000000       *
           * step 1: body_list = 100010001000000100100       *
           * step 2: body_list = 111122223333333444555       *
           * step 3: body_list = 012301230123456012012       *
           *         assuming that sh_body[j] = 0            *
           ***************************************************/

          n_bodies     -= nb;                                   //subtract from n_bodies number of bodies that have been extracted
          nl_pre       += nl;                                   //increase the number of leaves that where processed
          direct [tid] -= nb;                                   //subtract the number of extracted bodies in this pass
          prefix0[tid] = max(prefix0[tid] - nb, 0);             //same here, but do not let the number be negative (GT200 bug!?)
          n_direct     += nb;                                  //increase the number of bodies to be procssed

          while(n_direct >= DIM) 
          {
            n_direct -= DIM;

            const float4 posj  = body_pos[body_list[n_direct + tid]];
            const float4 hj    = body_h  [body_list[n_direct + tid]];
            sh_mass[tid] = posj.w;
            sh_pos [tid] = (float3){posj.x, posj.y, posj.z};
            sh_h   [tid] = (float3){hj  .x, hj  .y, hj  .z};  /* h, hinv, hflag */
            sh_jid [tid] = bodies_IDs[body_list[n_direct + tid]];
            __syncthreads();

#pragma unroll
            for (int j = 0; j < DIMx; j++)
            {
              direCount++;
              acc_i = body_body(
                  acc_i, 
                  pos_i, 
                  h_i,
                  sh_mass[offs + j], 
                  sh_pos [offs + j], 
                  sh_h   [offs + j],
                  body_i != sh_jid[offs + j]);
            }
            __syncthreads();
          }
        }
        direct[tid] = body_list[tid];
        __syncthreads();
#endif /* DIRECT */
      } //end of the tree level, proceed to the next one


      n_nodes1 += n_offset;
      if (n_offset > 0)
      { 
        nstack[ACCS<SHIFT>(n_stack1)] = nodes[tid];   n_stack1++; 
        if((n_stack1 - c_stack0) >= (LMEM_STACK_SIZE << SHIFT))
        {
          //We overwrote our current stack
          apprCount = -1; 
          return acc_i.to_float4();
        }
      }
      __syncthreads();


      /***
       **** --> copy nodes1 to nodes0: done by reassigning the pointers
       ***/
      n_nodes0    = n_nodes1;

      n_stack_pre = n_stack0;
      n_stack0    = n_stack1;

    }//end while   levels
  }//end for


  if(n_approx > 0)
  {

    if (tid < n_approx) {
      int address      = (approx[tid] << 1) + approx[tid];
#ifndef TEXTURES
      float4 monopole  = multipole_data[address    ];
      float4 octopole0 = multipole_data[address + 1];
      float4 octopole1 = multipole_data[address + 2];
#else
      float4 monopole  = tex1Dfetch(texMultipole, address);
      float4 octopole0 = tex1Dfetch(texMultipole, address + 1);
      float4 octopole1 = tex1Dfetch(texMultipole, address + 2);
#endif

      node_mon0[tid] = monopole.w;
      node_mon1[tid] = (float3){monopole.x,  monopole.y,  monopole.z};
      node_oct0[tid] = (float3){octopole0.x, octopole0.y, octopole0.z};
      node_oct1[tid] = (float3){octopole1.x, octopole1.y, octopole1.z};

    } else {
    
      //Set non-active memory locations to zero
      node_mon0[tid] = 0.0f;
      node_mon1[tid] = (float3){1.0e10f, 1.0e10f, 1.0e10f};
      node_oct0[tid] = (float3){0.0f, 0.0f, 0.0f};
      node_oct1[tid] = (float3){0.0f, 0.0f, 0.0f};
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < DIMx; i++)
    {
      apprCount++;
      acc_i = body_node(acc_i, pos_i,
          node_mon0[offs + i], node_mon1[offs + i],
          node_oct0[offs + i], node_oct1[offs + i]);
    }

    __syncthreads();
  } //if n_approx > 0

  if(n_direct > 0)
  {
    if (tid < n_direct) 
    {
      const float4 posj = body_pos[direct[tid]];
      const float4 hj   = body_h  [direct[tid]];
      sh_mass[tid] = posj.w;
      sh_pos [tid] = (float3){posj.x, posj.y, posj.z};
      sh_h   [tid] = (float3){hj  .x, hj  .y, hj  .z};  /* h, hinv, hflag */
      sh_jid [tid] = bodies_IDs[direct[tid]];

    } 
    else 
    {
      sh_mass[tid] = 0.0f;
      sh_pos [tid] = (float3){1.0e10f, 1.0e10f, 1.0e10f};
      sh_jid [tid] = -1;
      sh_h   [tid] = (float3){0.0f, 0.0f, 0.0f};
    }
    __syncthreads();

#pragma unroll
    for (int j = 0; j < DIMx; j++) 
      if ((sh_jid[offs + j] >= 0)) 
      {
        direCount++;
        acc_i = body_body(
            acc_i, 
            pos_i, 
            h_i,
            sh_mass[offs + j], 
            sh_pos [offs + j], 
            sh_h   [offs + j],
            body_i != sh_jid[offs + j]);
      }
    __syncthreads();
  }

  shForce[tid] = acc_i;
  __syncthreads();
  if (ty == 0)
    for (int i = 1; i < DIMy; i++) 
    {
      const int idx = (i << DIM2x) + tx;
      acc_i.ax  += shForce[idx].ax .to_float();
      acc_i.ay  += shForce[idx].ay .to_float();
      acc_i.az  += shForce[idx].az .to_float();
#ifdef _GPOTDP_
      acc_i.pot += shForce[idx].pot.to_float();
#else
      acc_i.pot += shForce[idx].pot;
#endif
    }
  __syncthreads();

  return acc_i.to_float4();
}


  extern "C" __global__ void
  __launch_bounds__(NTHREAD)
dev_approximate_gravity(
    const int n_active_groups,
    float eps2,
    uint2 node_begend,   
    int    *atomicValues,
    real4  *body_pos,                        
    real4  *body_h,
    float4 *acc_out,
    real4  *group_body_pos, 
    real4  *group_body_h,
    real   *ds2_out,
    int    *ngb_out,
    int    *active_inout,
    int2   *interactions,
    uint2  *group_list,
    real4  *multipole_data,
    float4  *boxSizeInfo,                        
    float4  *boxCenterInfo,                        
    int     *MEM_BUF,
    int     *bodies_IDs,
    int     *group_bodies_IDs) 
{


  const int blockDim2 = NTHREAD2;
  __shared__ int shmem[18*(1 << blockDim2)];
  //    __shared__ int shmem[24*(1 << blockDim2)]; is possible on FERMI
  //    int             lmem[LMEM_STACK_SIZE];



  /*********** check if this block is linked to a leaf **********/

  int bid = gridDim.x * blockIdx.y + blockIdx.x;

  while(true)
  {

    if(threadIdx.x == 0)
    {
      bid         = atomicAdd(&atomicValues[0], 1);
      shmem[0]    = bid;
    }
    __syncthreads();

    bid   = shmem[0];

    if (bid >= n_active_groups) return;


    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    //   volatile int *lmem = &MEM_BUF[blockIdx.x*LMEM_STACK_SIZE*blockDim.x + threadIdx.x*LMEM_STACK_SIZE];
    //   int *lmem = &MEM_BUF[blockIdx.x*LMEM_STACK_SIZE*blockDim.x + threadIdx.x*LMEM_STACK_SIZE];
    int *lmem = &MEM_BUF[blockIdx.x*LMEM_STACK_SIZE*blockDim.x];


    /*********** set necessary thread constants **********/

    //   real4 curGroupSize    = groupSizeInfo[active_groups[bid + grpOffset]];
    //   int   groupData       = __float_as_int(curGroupSize.w);
    //   uint body_i           =   groupData & CRITMASK;
    //   uint nb_i             = ((groupData & INVCMASK) >> CRITBIT) + 1;
    // 
    //   real4 group_pos       = groupCenterInfo[active_groups[bid + grpOffset]];

    //   if(tid == 0)
    //   printf("[%f %f %f %f ] \n [%f %f %f %f ] %d %d \n",
    //           curGroupSize.x, curGroupSize.y, curGroupSize.z, curGroupSize.w,
    //           group_pos.x, group_pos.y, group_pos.z, group_pos.w, body_i, nb_i);
    uint2 grpInfo = group_list[bid];
    uint body_i = grpInfo.x;
    uint nb_i   = (grpInfo.y - grpInfo.x) + 1;

    int DIM2x = 0;
    while (((nb_i - 1) >> DIM2x) > 0) DIM2x++;

    DIM2x     = max(DIM2x,4);
    int DIM2y = blockDim2 - DIM2x;

    int tx = tid & ((1 << DIM2x) - 1);
    int ty = tid >> DIM2x;

    body_i += tx%nb_i;
   
    float4 acc_i = {0.0f, 0.0f, 0.0f, 0.0f};

    const float4 pos_i = group_body_pos[body_i];
    const float4 h_i   = group_body_h  [body_i];

    real4 group_pos;
    real4 curGroupSize;

    computeGroupProps(group_pos, curGroupSize, pos_i, shmem);
    float group_eps = 0;        //This is disabled for the moment

    int ngb_i = -1;
    float ds2 = -1;

    int apprCount = 0;
    int direCount = 0;

    const int body_id = group_bodies_IDs[body_i];

#if 1
    acc_i = approximate_gravity<blockDim2, 0>(
        DIM2x, DIM2y, tid, tx, ty,
        body_id, pos_i, h_i, group_pos,
        eps2, node_begend,
        multipole_data, body_pos,
        shmem, lmem, 
        apprCount, direCount, boxSizeInfo, curGroupSize, boxCenterInfo,
        group_eps, 
        body_h,
        bodies_IDs);
#endif
#if 0
    if (apprCount < 0)
    {
      printf(" --------- Error:: body_i= %d \n", body_i);
    }
#else
    if(apprCount < 0)
    {

      //Try to get access to the big stack, only one block per time is allowed
      if(threadIdx.x == 0)
      {
        int res = atomicExch(&atomicValues[1], 1); //If the old value (res) is 0 we can go otherwise sleep
        int waitCounter  = 0;
        while(res != 0)
        {
          //Sleep
          for(int i=0; i < (1024); i++)
          {
            waitCounter += 1;
          }
          //Test again
          shmem[0] = waitCounter;
          res = atomicExch(&atomicValues[1], 1); 
        }
      }

      __syncthreads();

      lmem = &MEM_BUF[gridDim.x*LMEM_STACK_SIZE*blockDim.x];    //Use the extra large buffer
      apprCount = direCount = 0;
      acc_i = approximate_gravity<blockDim2, 8>( DIM2x, DIM2y, tid, tx, ty,
          body_id, pos_i, h_i, group_pos,
          eps2, node_begend,
          multipole_data, body_pos,
          shmem, lmem, 
          apprCount, direCount, boxSizeInfo, curGroupSize, boxCenterInfo,
          group_eps, 
          body_h,
          bodies_IDs);

      lmem = &MEM_BUF[blockIdx.x*LMEM_STACK_SIZE*blockDim.x]; //Back to normal location

      if(threadIdx.x == 0)
      {
        atomicExch(&atomicValues[1], 0); //Release the lock
      }
    }//end if apprCount < 0
#endif


    if (tid < nb_i) 
    {
      acc_out     [body_i] = acc_i;
      ngb_out     [body_i] = ngb_i;
      ds2_out     [body_i] = ds2;
      active_inout[body_i] = 1;
      interactions[body_i].x = apprCount;
      interactions[body_i].y = direCount ;
    }
  }     //end while
}


