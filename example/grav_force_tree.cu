#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include "libSequoia/include/sequoiaInterface.h"
#include "vector3.h"

struct Particle
{
  typedef std::vector<Particle> Vector;
  vec3   pos;
  double mass, h, hflag;
  Particle() {}
  Particle(const vec3 &_pos, double _mass, const double _h, const double _hf) :
    pos(_pos), mass(_mass), h(_h), hflag(_hf) {}
};

struct Force
{
  typedef std::vector<Force> Vector;
  vec3 acc;
  double pot;
  Force() {}
  Force(const vec3 &_acc, const double _pot = 0) : acc(_acc), pot(_pot) {}
};

struct SPHgrav_tree
{
  Particle::Vector ptcl;
  Force   ::Vector force;
  
  hipEvent_t start, stop;

  my_dev::context devContext;
    
  /* Input */
  my_dev::dev_mem<real4> j_bodies_pos;
  my_dev::dev_mem<real4> j_bodies_h  ;
  my_dev::dev_mem<int>   j_bodies_IDs;

  my_dev::dev_mem<real4> i_bodies_pos;
  my_dev::dev_mem<real4> i_bodies_h  ;
  my_dev::dev_mem<int>   i_bodies_IDs;

  /* Output */
  my_dev::dev_mem<real4> bodies_acc;
  my_dev::dev_mem<real>  bodies_ds2;
  my_dev::dev_mem<int>   bodies_ngb;



  int nibeg, ni;
  int ndevice;
  std::vector<bool> can_use_device;
  char *argv[1];
  bool init_context;
  SPHgrav_tree(const int rank = 0)
  {
    init_context = false;
    assert(hipGetDeviceCount(&ndevice) == 0);
    if (rank == 0)
      fprintf(stderr, " SPHgrav found %d CUDA devices \n", ndevice);
    assert(ndevice > 0);
    can_use_device = std::vector<bool>(ndevice, false);
    int no_supported = 0;
    for (int dev = 0; dev < ndevice; dev++)
    {
      hipDeviceProp_t p;
      assert(hipGetDeviceProperties(&p, dev) == hipSuccess);
      const bool supported = p.major > 1 || (p.major == 1 && p.minor >= 3);
      if (rank == 0)
        fprintf(stderr,"  Device= %d: %s computeMode= %d computeCapability= %d_%d  supported= %s\n", 
            dev, p.name, p.computeMode, p.major, p.minor, supported ? "YES" : "NO");
      no_supported += supported ? 1 : 0;
      can_use_device[dev] = true;
    }
    assert(no_supported > 0);

    argv[0] = new char[256];
    sprintf(argv[0], "./");
#if 1
    sprintf(argv[0], "./src/SPHgravtree_lib/libSequoia/");
#endif
  }
  ~SPHgrav_tree()
  {
    delete[] argv[0];
    sequoia_cleanup();
  }

  void initDevice(const int device, const double theta = 0.3)
  {
    assert(!init_context);
    init_context = true;
    assert(device < ndevice);
    assert(can_use_device[device]);
    fprintf(stderr, " >>> Setting device %d , theta= %g <<< \n", device, theta);
    devContext =  sequoia_init(argv, device, theta, 0.0);
   
    /* Input */ 
    j_bodies_pos.setContext(devContext);    //Bodies positions
    j_bodies_h  .setContext(devContext);    //Bodies range
    j_bodies_IDs.setContext(devContext);    //Bodies idx
    
    i_bodies_pos.setContext(devContext);    //Bodies positions
    i_bodies_h  .setContext(devContext);    //Bodies range
    i_bodies_IDs.setContext(devContext);    //Bodies idx
    
    /* Output */
    bodies_acc.setContext(devContext);    //Bodies Accelerations
    bodies_ds2.setContext(devContext);    //Bodies distance to nearest neighbour squared
    bodies_ngb.setContext(devContext);    //Bodies nearest neighbour
  }

  void first_half(const int nibeg, const int ni)
  {
    assert(init_context);
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    hipEventRecord( start, 0 );

    this->nibeg = nibeg;
    this->ni    = ni;
    force.resize(ni);

    int nj = ptcl.size();

    /* Input */
    j_bodies_pos.cmalloc(nj);    //Bodies positions
    j_bodies_h  .cmalloc(nj);    //Bodies range
    j_bodies_IDs.cmalloc(nj);    //Bodies idx

    //Jeroen changed this to size ni, since its possible
    //that ni is not as large as nj in parallel code
    i_bodies_pos.cmalloc(ni);    //Bodies positions
    i_bodies_h  .cmalloc(ni);    //Bodies range
    i_bodies_IDs.cmalloc(ni);    //Bodies idx

    /* Output */
    bodies_acc.cmalloc(ni);    //Bodies Accelerations
    bodies_ds2.cmalloc(ni);    //Bodies distance to nearest neighbour squared
    bodies_ngb.cmalloc(ni);    //Bodies nearest neighbour

    for (int i = 0; i < nj; i++)
    {
      j_bodies_pos[i].x = ptcl[i].pos.x;
      j_bodies_pos[i].y = ptcl[i].pos.y;
      j_bodies_pos[i].z = ptcl[i].pos.z;
      j_bodies_pos[i].w = ptcl[i].mass;

      j_bodies_h[i].x =     ptcl[i].h;
      j_bodies_h[i].y = 1.0/ptcl[i].h;
      j_bodies_h[i].z =     ptcl[i].hflag;

      j_bodies_IDs[i] = i;
    }

    j_bodies_pos.h2d();
    j_bodies_h  .h2d();
    j_bodies_IDs.h2d();

    for (int i = 0; i < ni; i++)
    {
      const int j = i + nibeg;

      i_bodies_pos[i].x = ptcl[j].pos.x; //Jeroen, first i was j, copy paste error
      i_bodies_pos[i].y = ptcl[j].pos.y;
      i_bodies_pos[i].z = ptcl[j].pos.z;
      i_bodies_pos[i].w = ptcl[j].mass;

      i_bodies_h[i].x =     ptcl[j].h;
      i_bodies_h[i].y = 1.0/ptcl[j].h;
      i_bodies_h[i].z =     ptcl[j].hflag;

      i_bodies_IDs[i] = j;
    }

    i_bodies_pos.h2d();
    i_bodies_h  .h2d();
    i_bodies_IDs.h2d();

    sequoia_setParticlesAndGetGravity(
        j_bodies_pos,      //Positions J-particles
        j_bodies_h,       
        j_bodies_IDs,      //Particle IDs J-particles
        nj,         //Number of J-particles
        i_bodies_pos,      //Positions I-particles (Can be the same or different than J-particles)
        i_bodies_h,       
        i_bodies_IDs,      //Particle IDs J-particles (Can be the same or different than J-particles)
        ni,                  //Number of I-particles (Can be the same or different than J-particles)
        true,              //Do we need to sort J-particles?
        true,              //Do we need to sort I-particles? Can be false if i-bodies are the same as j-bodies           
        bodies_acc,        //OUT  Accelerations for I-particles
        bodies_ds2,        //OUT  min distance squared for I-particles
        bodies_ngb);       //OUT  J-ID of the nearest neighbour for I-particles

  }

  float last_half()
  {
    bodies_acc.d2h();
    i_bodies_IDs.d2h();

    for (int i = 0; i < ni; i++)
    {
      const int idx = i_bodies_IDs[i];
      assert(idx >= nibeg);
      assert(idx < ni+nibeg);
      //force[idx] = Force(vec3(bodies_acc[i].x, bodies_acc[i].y, bodies_acc[i].z), bodies_acc[i].w);
      //Jeroen, idx-nibeg otherwise there is no valid memory location if nibeg > 0 and all kind of bad
      //things start to happen
      force[idx-nibeg] = Force(vec3(bodies_acc[i].x, bodies_acc[i].y, bodies_acc[i].z), bodies_acc[i].w);
    }

    hipEventRecord( stop, 0 );
    hipDeviceSynchronize();
    float elapsed_time_ms;
    hipEventElapsedTime( &elapsed_time_ms, start, stop );
    
    /* Input */
    j_bodies_pos.free_mem();
    j_bodies_h  .free_mem();
    j_bodies_IDs.free_mem();

    i_bodies_pos.free_mem();
    i_bodies_h  .free_mem();
    i_bodies_IDs.free_mem();

    /* Output */
    bodies_acc.free_mem();
    bodies_ds2.free_mem();
    bodies_ngb.free_mem();


    return elapsed_time_ms/1000.0;
  }
};

namespace SPHgrav
{
  SPHgrav_tree grav;

  void firsthalf_grav_force(
      const int nj, const int nibeg, const int ni,
      double *px, double *py, double *pz,
      double *mass, double *h2)
  {
    grav.ptcl.resize(nj);

    assert(nibeg >= 0);

    for (int i = 0; i < nj; i++)
    {
      assert(h2  [i] != 0.0);
      assert(mass[i] >  0.0);
      grav.ptcl[i] = Particle(vec3(px[i], py[i], pz[i]), mass[i], 
          std::sqrt(h2[i] > 0.0 ? h2[i] : -h2[i]),
          h2[i] > 0.0 ? 1.0 : 0.0);
    }

    grav.first_half(nibeg, ni);
  }

  void lasthalf_grav_force(double *ax, double *ay, double *az, double *pot)
  {
    const int ni  = grav.ni;
    const float dt = grav.last_half();

    for (int i = 0; i < ni; i++)
    {
      const int j = i + grav.nibeg;
      ax [j] = grav.force[i].acc.x;
      ay [j] = grav.force[i].acc.y;
      az [j] = grav.force[i].acc.z;
      pot[j] = grav.force[i].pot;
    }

#if 1
    fprintf(stderr, " >>> SPHgrav_lib took %g sec \n", dt);
#endif
  }

  void initDevice(const int device, const double theta)
  {
    grav.initDevice(device, theta);   /* opening angle, 0.3 or 0.4 should be safe */
  }
}

extern "C"
{
  void firsthalf_grav_forces_(
      int *n, int *n_lower, int *my_length, 
      double *px, double *py, double *pz,
      double *mass, double *range2, int *q_)
  {
    SPHgrav::firsthalf_grav_force(*n, *n_lower - 1, *my_length, px, py, pz, mass, range2);
  }

  void lasthalf_grav_forces_(
      int *n, 
      double *ax, double *ay, double *az,
      double *pot,int *myrank) 
  {
    SPHgrav::lasthalf_grav_force(ax, ay, az, pot);
  }

  void gpu_init_dev_(int *myrank, double *theta)
  {
    SPHgrav::initDevice(*myrank, *theta);
  }
}

